#include "gpu_runtime.h"

int DLGpuBatchMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                             const DLArrayHandle matB, bool transposeB,
                             DLArrayHandle matC,
                             DLStreamHandle stream_handle = NULL) {
    assert(matA->ndim == matB->ndim);
    assert(matA->ndim == matC->ndim);

    int dev_id = (matA->ctx).device_id;
    cublas_init(dev_id, stream_handle);

    float one = 1.0f;
    float zero = 0.0f;

    int ndim = matA->ndim;
    int m = matC->shape[ndim - 1];
    int n = matC->shape[ndim - 2];
    int k = transposeA ? matA->shape[ndim - 2] : matA->shape[ndim - 1];
    long long int strideA = matA->shape[ndim - 2] * matA->shape[ndim - 1];
    long long int strideB = matB->shape[ndim - 2] * matB->shape[ndim - 1];
    long long int strideC = matC->shape[ndim - 2] * matC->shape[ndim - 1];

    int batchCount = 1;
    for (int i = 0; i < ndim - 2; ++i) {
        assert(matA->shape[i] == matB->shape[i]);
        assert(matA->shape[i] == matC->shape[i]);
        batchCount *= matA->shape[i];
    }

    hipDataType data_type = HIP_R_32F;
    hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
    hipblasStatus_t res = hipblasGemmStridedBatchedEx(
        cublas_map[dev_id], transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &one,
        (const float *)matB->data, data_type, !transposeB ? m : k, strideB,
        (const float *)matA->data, data_type, !transposeA ? k : n, strideA, &zero,
        (float *)matC->data, data_type, m, strideC, batchCount, data_type, algo);
    assert(res == HIPBLAS_STATUS_SUCCESS);
    return 0;
}
