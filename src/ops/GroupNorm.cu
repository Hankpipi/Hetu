#include "hip/hip_runtime.h"
#include "gpu_reduce.h"

__global__ void group_norm_forward(const float *x,
                                   const float *scale,
                                   const float *bias,
                                   float* y,
                                   float *mean, float *var,
                                   const float eps, const int N, const int M, const int HW, const int C) {
    __shared__ float var_share;
    __shared__ float mean_share;
    __shared__ float shared_var[32];
    __shared__ float shared_mean[32];

    for(int row = blockIdx.x; row < N; row += gridDim.x) {
        int begin = row * M + threadIdx.x;
        int end = (row + 1) * M;

        float mean_thread = 0, var_thread = 0;
        for (int i = begin; i < end; i += blockDim.x) {
            mean_thread += x[i];
            var_thread += x[i] * x[i];
        }

        BlockReduceSum(mean_thread, shared_mean);
        BlockReduceSum(var_thread, shared_var);
        if (threadIdx.x == 0) {
            mean[row] = mean_share = mean_thread / M;
            var_share = var_thread / M  - mean_share * mean_share;
            if (var_share < 0) var_share = 0;
            var[row] = var_share;
        }
        __syncthreads();    

        mean_thread = mean_share;
        var_thread = var_share;
        float tmp = 1.0f / sqrtf(var_thread + eps);
        for (int i = begin; i < end; i += blockDim.x) {
            int cid = i / HW % C;
            y[i] = (x[i] - mean_thread) * tmp * scale[cid] + bias[cid];
        }
    }
}

int DLGpuGroupNormalization(const DLArrayHandle in_arr,
                            const DLArrayHandle ln_scale,
                            const DLArrayHandle ln_bias, 
                            int num_groups, DLArrayHandle mean_arr,
                            DLArrayHandle var_arr, DLArrayHandle out_arr,
                            float eps, DLStreamHandle stream_handle) {
    int ndim = in_arr->ndim;
    int C = 1, HW = 1;
    int B = in_arr->shape[0] * num_groups;
    for(int i = 1; i < ndim; ++i)
        C *= in_arr->shape[i];
    HW = C / in_arr->shape[1];
    C /= num_groups;

    dim3 blocks;
    dim3 threads;
    int rows_per_block = (C <= 8192? 2: 1);
    blocks.x = (B + rows_per_block - 1) / rows_per_block;
    threads.x = GetThreadNum(C);
    if (stream_handle)
        group_norm_forward<<<blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
                (const float *)in_arr->data, (const float *)ln_scale->data,
                (const float *)ln_bias->data, (float *)out_arr->data,
                (float *)mean_arr->data, (float *)var_arr->data, eps, B, C, HW, in_arr->shape[1]);
    else
        group_norm_forward<<<blocks, threads, 0>>>(
                (const float *)in_arr->data, (const float *)ln_scale->data,
                (const float *)ln_bias->data, (float *)out_arr->data,
                (float *)mean_arr->data, (float *)var_arr->data, eps, B, C, HW, in_arr->shape[1]);
    return 0;
}
